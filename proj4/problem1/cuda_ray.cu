
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <chrono>
#include <thread>

#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere {
	float   r, b, g;
	float   radius;
	float   x, y, z;
};


__global__
void kernel(Sphere* s, unsigned char* ptr)
{
    int x,y;
	x = blockIdx.x/8; // in range 0 to 2048, will be fixed for 2048 iterations (during this time, y will increment)
	y = threadIdx.x + 256*(blockIdx.x%8); //in range 0 to 2048, 256 is total number of threads, for the first block, it's just the thread ID

	int offset = x + y * 2048;
	float ox = (x - 2048 / 2);
	float oy = (y - 2048 / 2);

	//printf("x:%d, y:%d, ox:%f, oy:%f\n",x,y,ox,oy);

	float r = 0, g = 0, b = 0;
	float   maxz = -2e10f;
	for (int i = 0; i < 20; i++) {

		float   n, t;
        float dx = ox - s[i].x;
        float dy = oy - s[i].y;
        if (dx * dx + dy * dy < s[i].radius * s[i].radius) {
            float dz = sqrtf(s[i].radius * s[i].radius - dx * dx - dy * dy);
            n = dz / sqrtf(s[i].radius * s[i].radius);
            t = dz + s[i].z;
        }
        else{
            t = -2e10f;

        }
		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	}
	ptr[offset * 4 + 0] = (int)(r * 255);
	ptr[offset * 4 + 1] = (int)(g * 255);
	ptr[offset * 4 + 2] = (int)(b * 255);
	ptr[offset * 4 + 3] = 255;

}


void ppm_write(unsigned char* bitmap, int xdim, int ydim, FILE* fp)
{
	int i, x, y;
	fprintf(fp, "P3\n");
	fprintf(fp, "%d %d\n", xdim, ydim);
	fprintf(fp, "255\n");
	for (y = 0;y < ydim;y++) {
		for (x = 0;x < xdim;x++) {
			i = x + y * xdim;
			fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
		}
		fprintf(fp, "\n");
	}
}


int main(int argc, char* argv[])
{
	unsigned char* bitmap;
	Sphere *dev_s; //for spheres in the GPU
	unsigned char *dev_bitmap; //for result in GPU
	srand(time(NULL));

	if (argc != 2) {
		printf("> a.out [filename.ppm]\n");
		printf("for example, '> a.out result.ppm'.\n");
		exit(0);
	}

	FILE* fp = fopen(argv[1], "w");

	Sphere* temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
	for (int i = 0; i < SPHERES; i++) {
		temp_s[i].r = rnd(1.0f);
		temp_s[i].g = rnd(1.0f);
		temp_s[i].b = rnd(1.0f);
		temp_s[i].x = rnd(2000.0f) - 1000;
		temp_s[i].y = rnd(2000.0f) - 1000;
		temp_s[i].z = rnd(2000.0f) - 1000;
		temp_s[i].radius = rnd(200.0f) + 40;
	}
	hipMalloc((void **)&dev_s, sizeof(Sphere) * SPHERES); //allocate place for sphere in GPU
	hipMemcpy(dev_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice); //copy CPU Spheres into GPU

	bitmap = (unsigned char*)malloc(sizeof(unsigned char) * DIM * DIM * 4);
	hipMalloc((void **)&dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4); //allocate place for result in GPU
	hipMemcpy(dev_bitmap, bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyHostToDevice); //copy CPU bitmap into GPU

	printf("Execution begins...\n");
	clock_t start_time = clock();
    int block_size = 256; //256 threads by block
    int num_block = (DIM*DIM + block_size - 1)/block_size; //16384 blocks
    kernel<<<num_block,block_size>>>(dev_s, dev_bitmap);
    hipDeviceSynchronize(); //CPU waits for GPU

	hipMemcpy(bitmap, dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost); //copy back the result from GPU to CPU

	clock_t end_time = clock(); //end of cuda computation
	ppm_write(bitmap, DIM, DIM, fp); //write result in file

	// free allocation
	fclose(fp);
	free(bitmap);
	free(temp_s);
	hipFree(temp_s);

	clock_t diff_time = end_time - start_time;
	printf("\t Execution time: %d ms. \n", diff_time);
	printf("Execution ended.\n");

	return 0;
}
