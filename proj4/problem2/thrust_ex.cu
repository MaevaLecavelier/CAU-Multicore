#include "hip/hip_runtime.h"
// Maeva Lecavelier - 50191580
// I choose exercice (b), integral approximation

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <stdio.h>
#include <time.h>
#include <iostream>


#define NUM_STEPS 200000
#define STEP 1/NUM_STEPS

struct integral
{
    __host__ __device__
        float operator() (int xi) const {
            float x;
            x = (float)xi * STEP;
            //printf("%f\n", (float)STEP * (4/(1+x*x)));
            return (float)STEP * (4/(1+x*x));
        }
};

int main(){
    printf("Execution begins...\n");
    clock_t start_time = clock();
    thrust::device_vector<float> index(NUM_STEPS+1, 1); //array with index
    thrust::device_vector<float> res(NUM_STEPS+1, 1); // array with rectangle area
    thrust::sequence(index.begin(), index.end()); //fill array from 0 to NUM_STEPS
    thrust::transform(index.begin(), index.end(), res.begin(), integral()); //res[0] = f(O)*STEP = area of one rectangle
    float sum = thrust::reduce(res.begin(), res.end()); //we add all little rectangle areas. The result is the integral
    clock_t end_time = clock();
    printf("Execution ends.\n");
    printf("Execution time: %d ms. \n", end_time - start_time);
    printf("pi=%f\n", sum); //print result

    return 0;
}
